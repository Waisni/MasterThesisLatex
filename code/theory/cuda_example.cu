
#include <hip/hip_runtime.h>
// This cuda kernel adds two vectors together and saves it into a third one 
__global__ void vectorAdd(const int* srcA, const int* srcB, const int vecSize, int* dst) {
    // Get the id of the current thread
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ignore all threads with an id larger or equal to vecSize to avoid running out of bound
    if (idx < vecSize) {
        // Add vectors srcA and arcB together and write it into the vector dst
        dst[idx] = srcA[idx] + srcB[idx];
    }
}
